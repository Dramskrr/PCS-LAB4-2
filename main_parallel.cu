//#include <limits.h>
//#include <stdint.h>
//#include <asm-generic/errno.h>
#include <cstdlib>
#include <stdio.h>
#include <stdlib.h>
//#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
//#include <stdbool.h>

const long int DEFAULT_ARRAY_SIZE = 134217728;
const int DEFAULT_RUNS = 2;
const int DEFAULT_THREADS = 256;
const int DEFAULT_BLOCKS = 8;

__global__ void BitonicSortStep(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

float* CreateArray( const int SIZE) {
    float* float_array = (float*) malloc(sizeof(float) * SIZE);
    for (int i = 0; i < SIZE; i++) {
        float_array[i] = rand()%100;
    }
    return float_array;
}

void PrintArray(const int* array, const int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        printf("%d ",array[i]);
    }
    printf("\n");
}

long int GetEnvArraySize() {
    char* array_size_char = getenv("ARRAY_SIZE");
    long int array_size_int = DEFAULT_ARRAY_SIZE;
    if (array_size_char != NULL) {
        array_size_int = atoi(array_size_char);
    } else {
        printf(
            "Переменная среды ARRAY_SIZE не получена, "
            "используем значение по умолчанию: %ld \n", DEFAULT_ARRAY_SIZE
        );
    }
    return array_size_int;
}

int GetEnvThreads() {
    char* thread_char = getenv("THREADS");
    int thread_int = DEFAULT_THREADS;
    if (thread_char != NULL) {
        thread_int = atoi(thread_char);
    } else {
        printf(
            "Переменная среды THREADS не получена, "
            "используем значение по умолчанию: %d \n", DEFAULT_THREADS
        );
    }
    return thread_int;
}

// int GetEnvBlocks() {
//     char* block_char = getenv("BLOCKS");
//     int block_int = DEFAULT_BLOCKS;
//     if (block_char != NULL) {
//         block_int = atoi(block_char);
//     } else {
//         printf(
//             "Переменная среды BLOCKS не получена, "
//             "используем значение по умолчанию: %d \n", DEFAULT_BLOCKS
//         );
//     }
//     return block_int;
// }

int GetEnvRuns() {
    char* runs_char = getenv("RUNS");
    int runs_int = DEFAULT_RUNS;
    if (runs_char != NULL) {
        runs_int = atoi(runs_char);
    } else {
        printf(
            "Переменная среды RUNS не получена, "
            "используем значение по умолчанию: %d \n", DEFAULT_RUNS
        );
    }
    return runs_int;
}

void CheckCudaError(hipError_t err){
    if (err != hipSuccess) {
        fprintf(stderr, "Fail (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void PrintArray(const float* array, const int SIZE) {
    for (int i = 0; i < SIZE; i++) {
        printf("%f ",array[i]);
    }
    printf("\n");
}

void CheckSort(float *array, const int SIZE){
    for (int i = 1; i < SIZE-1; i++){
        if (array[i] > array[i+1]){
            printf("Сортировка неверная!");
            return;
        }
    }
    printf("Сортировка верная!\n");
}

int main(int argc, char** argv) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    srand(time(0));
    //srand(1);
    const long int ARRAY_SIZE = GetEnvArraySize();
    const int RUNS = GetEnvRuns();
    const int THREADS = GetEnvThreads();
    //const int BLOCKS = GetEnvBlocks();

    if ((ARRAY_SIZE & (ARRAY_SIZE - 1)) != 0){
        printf("Размер массива не является степенью 2!\n");
        exit(EXIT_FAILURE);
    }
    const int BLOCKS = (ARRAY_SIZE / THREADS);

    printf("\n\nПараллельная программа\n");
    printf("Размер массива: %ld\n", ARRAY_SIZE);
    printf("Выполнений: %d\n", RUNS);
    printf("Потоков в блоке: %d\n", THREADS);
    printf("Блоков (ДЛЯ ДАННОГО ЗАДАНИЯ НАСТРОЙКА КОЛ-ВА БЛОКОВ ИГНОРИРУЕТСЯ,\n\
            ПРОГРАММА САМА ВЫСЧИТАЛА НУЖНОЕ КОЛИЧЕСТВО БЛОКОВ НА ОСНОВЕ КОЛ-ВА ПОТОКОВ): %d\n", BLOCKS);
    
    // Таймер
    struct timespec begin, end;
    double exec_time = 0.0;
    double data_allocation_time = 0.0;

    // Цикл выполнения задачи и подсчёта времени её выполнения
    for (int i = 0; i < RUNS; i++) {

        // Массив хоста с данными
        float* host_float_array = NULL;
        host_float_array = CreateArray(ARRAY_SIZE);

        clock_gettime(CLOCK_REALTIME, &begin); // Начало таймера

        // Выделение глобальной памяти под массив, который будет передан GPU
        float* device_float_array = NULL;
        err = hipMalloc(&device_float_array, ARRAY_SIZE * sizeof(float));
        CheckCudaError(err);
        //printf("Глоб массив выделен\n");
        
        //Копирование массива в GPU
        err = hipMemcpy(device_float_array,
                         host_float_array,
                         ARRAY_SIZE * sizeof(float),
                         hipMemcpyHostToDevice
                        );
        CheckCudaError(err);
        //printf("Глоб массив скопирован\n");

        clock_gettime(CLOCK_REALTIME, &end); // Конец таймера
        data_allocation_time += (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_nsec - begin.tv_nsec)/1e9;
        clock_gettime(CLOCK_REALTIME, &begin); // Начало таймера
        
        // Выполнение задачи
        // CUDA код позаимствован из
        // https://gist.github.com/mre/1392067
        int j = 0;
        int k = 0;
        /* Major step */
        for (k = 2; k <= ARRAY_SIZE; k <<= 1) {
            /* Minor step */
            for (j=k>>1; j>0; j=j>>1) {
                BitonicSortStep<<<BLOCKS, THREADS>>>(device_float_array, j, k);
            }
        }
        hipDeviceSynchronize();
        err = hipGetLastError();
        CheckCudaError(err);
        //printf("Задача выполнена\n");

        clock_gettime(CLOCK_REALTIME, &end); // Конец таймера
        exec_time += (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_nsec - begin.tv_nsec)/1e9;
        clock_gettime(CLOCK_REALTIME, &begin); // Начало таймера

        // Берём результат от GPU
        err = hipMemcpy(host_float_array,
                         device_float_array,
                         ARRAY_SIZE * sizeof(float),
                         hipMemcpyDeviceToHost
                        );
        CheckCudaError(err);
        //printf("Результат получен\n");
        
        // Освобождаем глобальную память GPU
        err = hipFree(device_float_array);
        CheckCudaError(err);
        //printf("Память очищена\n");

        clock_gettime(CLOCK_REALTIME, &end); // Конец таймера
        data_allocation_time += (double)(end.tv_sec - begin.tv_sec) + (double)(end.tv_nsec - begin.tv_nsec)/1e9;
        
        CheckSort(host_float_array, ARRAY_SIZE);

        free(host_float_array);
    }

    double mean_data_alloc_time = data_allocation_time / RUNS;
    double mean_exec_time = exec_time / RUNS;
    printf("Общее время выделения памяти, передачи данных и финального счёта: %f сек. \n", data_allocation_time);
    printf("Среднее время выделения памяти передачи данных и финального счёта: %f сек. \n\n", mean_data_alloc_time);
    printf("Общее время выполнения кода на GPU: %f сек. \n", exec_time);
    printf("Среднее время выполнения кода на GPU: %f сек. \n\n", mean_exec_time );
    printf("Общее время выполнения: %f сек. \n", exec_time + data_allocation_time);
    printf("Среднее время выполнения: %f сек.", mean_exec_time + mean_data_alloc_time);

    return 0;
}
